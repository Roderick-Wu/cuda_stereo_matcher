// vector_add_test.cu
// Minimal CUDA vector add test: compiles to an executable that
// performs a GPU vector addition and verifies the result on the host.

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

static inline void checkCuda(hipError_t e, const char* msg) {
    if (e != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(e));
        exit(1);
    }
}

__global__ void vecAddKernel(const float* a, const float* b, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) out[idx] = a[idx] + b[idx];
}

int main() {
    const int N = 1 << 16; // 65536 elements (small, fast)
    const size_t bytes = N * sizeof(float);

    float* h_a = (float*)malloc(bytes);
    float* h_b = (float*)malloc(bytes);
    float* h_out = (float*)malloc(bytes);
    if (!h_a || !h_b || !h_out) { fprintf(stderr, "Host alloc failed\n"); return 2; }

    for (int i = 0; i < N; ++i) { h_a[i] = (float)i; h_b[i] = (float)(2*i); }

    float *d_a=nullptr, *d_b=nullptr, *d_out=nullptr;
    checkCuda(hipMalloc((void**)&d_a, bytes), "hipMalloc d_a");
    checkCuda(hipMalloc((void**)&d_b, bytes), "hipMalloc d_b");
    checkCuda(hipMalloc((void**)&d_out, bytes), "hipMalloc d_out");

    checkCuda(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), "hipMemcpy h->d a");
    checkCuda(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice), "hipMemcpy h->d b");

    const int block = 256;
    const int grid = (N + block - 1) / block;
    vecAddKernel<<<grid, block>>>(d_a, d_b, d_out, N);
    checkCuda(hipGetLastError(), "kernel launch");
    checkCuda(hipDeviceSynchronize(), "hipDeviceSynchronize");

    checkCuda(hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost), "hipMemcpy d->h out");

    // verify
    for (int i = 0; i < N; ++i) {
        float expect = h_a[i] + h_b[i];
        if (h_out[i] != expect) {
            fprintf(stderr, "Mismatch at %d: got %f expected %f\n", i, h_out[i], expect);
            return 3;
        }
    }

    printf("vector_add_test: SUCCESS (%d elements)\n", N);

    hipFree(d_a); hipFree(d_b); hipFree(d_out);
    free(h_a); free(h_b); free(h_out);
    return 0;
}
