#include "hip/hip_runtime.h"
// vecadd_lib.cu
// Exports a simple C-callable API to perform vector addition on the GPU.

#include <hip/hip_runtime.h>
#include <cstdio>
#include "../include/vecadd.h"

static inline void checkCuda(hipError_t e, const char* msg) {
    if (e != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(e));
    }
}

__global__ void vecAddKernel(const float* a, const float* b, float* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) out[idx] = a[idx] + b[idx];
}

// Exported C API. Returns 0 on success, non-zero on failure.
extern "C" VECADD_API int vecAdd_host(const float* a_host, const float* b_host, float* out_host, int n) {
    if (!a_host || !b_host || !out_host || n <= 0) return 1;

    const size_t bytes = size_t(n) * sizeof(float);
    float *d_a = nullptr, *d_b = nullptr, *d_out = nullptr;

    hipError_t err = hipMalloc((void**)&d_a, bytes);
    if (err != hipSuccess) { checkCuda(err, "hipMalloc d_a"); return 2; }
    err = hipMalloc((void**)&d_b, bytes);
    if (err != hipSuccess) { checkCuda(err, "hipMalloc d_b"); hipFree(d_a); return 3; }
    err = hipMalloc((void**)&d_out, bytes);
    if (err != hipSuccess) { checkCuda(err, "hipMalloc d_out"); hipFree(d_a); hipFree(d_b); return 4; }

    err = hipMemcpy(d_a, a_host, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) { checkCuda(err, "hipMemcpy a"); goto cleanup; }
    err = hipMemcpy(d_b, b_host, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) { checkCuda(err, "hipMemcpy b"); goto cleanup; }

    const int block = 256;
    const int grid = (n + block - 1) / block;
    vecAddKernel<<<grid, block>>>(d_a, d_b, d_out, n);
    err = hipGetLastError();
    if (err != hipSuccess) { checkCuda(err, "kernel launch"); goto cleanup; }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) { checkCuda(err, "hipDeviceSynchronize"); goto cleanup; }

    err = hipMemcpy(out_host, d_out, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { checkCuda(err, "hipMemcpy out"); goto cleanup; }

    // success
    hipFree(d_a); hipFree(d_b); hipFree(d_out);
    return 0;

cleanup:
    if (d_a) hipFree(d_a);
    if (d_b) hipFree(d_b);
    if (d_out) hipFree(d_out);
    return 5;
}
